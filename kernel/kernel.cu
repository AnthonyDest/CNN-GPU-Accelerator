// Very minimal skeleton for the kernel


#include <hip/hip_runtime.h>
#include <stdio.h>


#define INPUT_DIM 100
#define FILTER_DIM 5
#define CONV_OUT_DIM 20
#define GRID_SIZE 100
#define NUM_FILTERS 10
#define FILTER_SIZE 5

extern "C" __global__ void ConvolutionLayer(const double input_data[GRID_SIZE][GRID_SIZE], const double filters[NUM_FILTERS][FILTER_SIZE][FILTER_SIZE], double output_data[10][20][20]) {
    int filter_index = blockIdx.x; // Index of the convolution filter

    int row = threadIdx.y; 
    int col = threadIdx.x; 

    for (int row_i = 0; row_i <5; row_i ++){
        for(int col_j = 0; col_j <5; col_j ++){
            output_data[filter_index][row][col] += input_data[row*5 + row_i][col*5 + col_j] * filters[filter_index][row_i][col_j];
        }
    }
}


extern "C" __global__ void relu_layer_kernel(double* conv_out, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_elements) {
        if (conv_out[idx] < 0.0f) {
            conv_out[idx] = 0.0f;
        }
    }
}

extern "C" __global__ void output_layer_kernel(const double *flat_input, const double weights[10][4000], double* output) {
    int neuron_offset = blockIdx.x;
        double sum = 0.0;
        for (int i = 0; i < 4000; i++) {
            sum += flat_input[i] * weights[neuron_offset][i];
        }
        output[neuron_offset] = sum;
}

extern "C" __global__ void output_layer_fast(const double input_A[4000], const double input_B[10][4000],  double output[10]) {   
    int neuron_idx = blockIdx.x; // 10
    int thread_idx = threadIdx.x; // 200

    __shared__ double first_array[10][200]; // Declaring first_array as shared memory

    // compute 20 dot products per thread
    for (int i = 0; i < 20; i++) {
        int input_idx = thread_idx * 20 + i;

        if (input_idx < 4000){
            first_array[neuron_idx][thread_idx] += input_A[input_idx] * input_B[neuron_idx][input_idx];
        }
    }   

    // waits for above to finish as kernel recognizes shared threads,

    // sum the partial dot products
    double sum = 0.0;
    for (int i = 0; i < 200; i++) {
        sum += first_array[neuron_idx][i];
    }

    output[neuron_idx] = sum;

    __syncthreads();
    // Resetting shared memory to zero
    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 200; j++) {
            first_array[i][j] = 0.0;
        }
    }
}
